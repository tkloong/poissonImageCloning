#include "hip/hip_runtime.h"
#include "../header/common.h"
#include "../header/poissonImageCloning.h"
#include <cstdio>
#include <time.h>

__global__ void PoissonImageEdit(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox,
	int stride
)
{
#define BOUND(x, h, w) \
	(((x) >= 0) && ((x) < (h)*(w)*3))
#define BOUND_MASK(x, h, w) \
	(((x) >= 0) && ((x) < (h)*(w))) 

	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;

	const int pt = (y*stride)*wt + x * stride;
	const int pb = (y*stride+oy)*wb + (x*stride+ox);

	int numberNeighbor = 0;
	int tNeighbors[4] = {-stride, stride, -stride*wt, stride*wt};
	bool tWeights[4] = {0, 0, 0, 0};

	// Out of range
	if (x*stride >= wt || y*stride >= ht) return;

	// Calculate number of neighbors
	for (int i=0; i<4; ++i) {
		if ( BOUND_MASK(pt + tNeighbors[i], ht, wt) && *(mask + pt + tNeighbors[i]) > 127.0f) {
			tWeights[i] = 1;
			numberNeighbor++;
		}
	}
	
	// All neighbors are black pixel out
	if (numberNeighbor == 0) return;

	float t0, t1, t2, t3, t4;
	float b1, b2, b3, b4;
	float o1, o2, o3, o4;
	double prev[3];
	double error[3];
	double totalError = 999.0;

	// Boundary
	for (int n=0; n<200 && totalError > 0.005; ++n) {
		memset(prev, 0, 3*sizeof(double));
		totalError = 0;
		if (numberNeighbor < 4 && *(mask + pt) > 127.0f ) {
			for (int i=0; i<3; ++i) {
				b1 = BOUND(pb-stride, hb, wb) ? *(background + (pb - stride)*3 + i) : 0;
				b2 = BOUND(pb+stride, hb, wb) ? *(background + (pb + stride)*3 + i) : 0;
				b3 = BOUND(pb-stride*wb, hb, wb) ? *(background + (pb - stride*wb)*3 + i) : 0;
				b4 = BOUND(pb+stride*wb, hb, wb) ? *(background + (pb + stride*wb)*3 + i) : 0;

				*(output + (pb)*3 + i) = ((!tWeights[0])*b1 + (!tWeights[1])*b2 + (!tWeights[2])*b3 
						+ (!tWeights[3])*b4)/(4-numberNeighbor);
			}
			for (int i=0; i<3; ++i) {
				totalError += error[i] * error[i];
			}
		}
		// Interior
		else if (*(mask + pt) > 127.0f){
			for (int i=0; i<3; ++i) {
				t0 = BOUND(pt, ht, wt) ? *(target + (pt)*3 + i) : 0;
				t1 = BOUND(pt-stride, ht, wt) ? *(target + (pt - stride)*3 + i) : 0;
				t2 = BOUND(pt+stride, ht, wt) ? *(target + (pt + stride)*3 + i) : 0;
				t3 = BOUND(pt-stride*wt, ht, wt) ? *(target + (pt - stride*wt)*3 + i) : 0;
				t4 = BOUND(pt+stride*wt, ht, wt) ? *(target + (pt + stride*wt)*3 + i) : 0;

				b1 = BOUND(pb-stride, hb, wb) ? *(background + (pb - stride)*3 + i) : 0;
				b2 = BOUND(pb+stride, hb, wb) ? *(background + (pb + stride)*3 + i) : 0;
				b3 = BOUND(pb-stride*wb, hb, wb) ? *(background + (pb - stride*wb)*3 + i) : 0;
				b4 = BOUND(pb+stride*wb, hb, wb) ? *(background + (pb + stride*wb)*3 + i) : 0;

				o1 = BOUND(pb-stride, hb, wb) ? *(output + (pb - stride)*3 + i) : 0;
				o2 = BOUND(pb+stride, hb, wb) ? *(output + (pb + stride)*3 + i) : 0;
				o3 = BOUND(pb-stride*wb, hb, wb) ? *(output + (pb - stride*wb)*3 + i) : 0;
				o4 = BOUND(pb+stride*wb, hb, wb) ? *(output + (pb + stride*wb)*3 + i) : 0;

				error[i] = prev[i] - *(output + pb*3 + i);
				*(output + (pb)*3 + i) = ((4*t0 - (t1 + t2 + t3 + t4)) + (o1 + o2 + o3 + o4))/4;
				prev[i] = *(output + pb*3 + i);
			}
			for (int i=0; i<3; ++i) {
				totalError += error[i] * error[i];
			}
		}
	}
}

__global__ void scaleUp(
		const float *mask,
		float *output, 
		const int wb, const int hb, const int wt, const int ht,
		const int oy, const int ox,
		int stride
		)
{
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;

	const int pt = y*wt + x;
	const int pb = (y+oy)*wb + (x+ox);

	if (x >= wt || y >= ht || *(mask + pt) < 127.0f) return;

	int pr = ((y/stride)*stride+oy)*wb + ((x/stride)*stride+ox);
	for (int i=0; i<3; i++)
		*(output + pb*3 + i) = *(output + pr*3 + i);
}

void PoissonImageCloning(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
	hipMemcpy(output, background, wb*hb*sizeof(float)*3, hipMemcpyDeviceToDevice);

	SimpleClone<<<dim3(CeilDiv(wt,32), CeilDiv(ht,16)), dim3(32,16)>>>(
			background, target, mask, output,
			wb, hb, wt, ht, oy, ox
			);

	clock_t t = clock();

	for (int scale=16; scale>1; scale>>=1) {
		PoissonImageEdit<<<dim3(CeilDiv(wt,32), CeilDiv(ht,16)), dim3(32,16)>>>(
				background, target, mask, output,
				wb, hb, wt, ht, oy, ox, scale
				);
		scaleUp<<<dim3(CeilDiv(wt,32), CeilDiv(ht,16)), dim3(32,16)>>>(
				mask, output, wb, hb, wt, ht, oy, ox, scale
				);
	}

	PoissonImageEdit<<<dim3(CeilDiv(wt,32), CeilDiv(ht,16)), dim3(32,16)>>>(
			background, target, mask, output,
			wb, hb, wt, ht, oy, ox, 1
			);

	printf("Time spent: %lf\n", (double)(clock() - t)/CLOCKS_PER_SEC);
}
